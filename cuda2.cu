
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


// CUDA kernel function to add elements of an array on the GPU
__global__
void add(int n, float *x, float *y){

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < n; i+= stride){
        y[i] = x[i] + y[i];
    }
}

int main(void){

    float *x;
    float *y;
    int N = 1<<20; 

    // Allocating memory 
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initializing arrays on host device
    for (int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // now running on 256 threads
    add<<< 1, 256>>>(N, x, y);

    // Waiting for GPU to finish
    hipDeviceSynchronize();

    float mError = 0.0f;
    // Checking for errors 
    for (int i = 0; i < N; i++){
        mError = fmax(mError, fabs(y[i] -3.0f));
    }

    std::cout << "Max Error: " << mError << std::endl;

    // free memory 
    hipFree(x);
    hipFree(y);
    

    return 0;
}